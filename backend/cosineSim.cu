#include "hip/hip_runtime.h"
#include<iostream>
#include<hip/hip_runtime.h>

#define nThreads 512
#define nBlocks 49
#define N nThreads*nBlocks

__global__ void dot(double *d_a,double *d_b,double *d_o){

    int t_id = threadIdx.x;
    int g_id = blockDim.x * blockIdx.x + threadIdx.x;
    int incr = gridDim.x * blockIdx.x;

    __shared__ double cache[nThreads];
    double temp=0.0;

    while(g_id < N){
        temp += d_a[g_id] * d_b[g_id];
        g_id += incr;
    }
    cache[t_id] = temp;
    __syncthreads();

    uint offset = blockDim.x/2;
    while(offset != 0){
        if(t_id < offset){
            cache[t_id] += cache[t_id+offset];
        }
        offset/=2;
        __syncthreads();
    }
    if(t_id==0){
        *d_o=cache[0];
    }   
}
 

__global__ void magnitude(double *d_a,double *d_o) {
   uint t_id = threadIdx.x;
   uint g_id = blockDim.x * blockIdx.x + threadIdx.x;
   uint incr = gridDim.x * blockIdx.x;

   __shared__ double sq[nThreads];
   double temp = 0.0;
   while(g_id<N){
        temp = pow(d_a[g_id],2);
        g_id += incr;
   }
   sq[t_id]= temp;
   __syncthreads();

   uint offset = blockDim.x/2;
   while(offset != 0){
       if(t_id < offset){
           sq[t_id] += sq[t_id+offset];
       }
       offset /= 2;
       __syncthreads();
   }
   if(t_id ==0){
       sq[0] = sqrt(sq[0]);
        *d_o = sq[0];
   }
}

int main(void)
{   
    double result=0.0;

    double a[] = {1.,2.,3.};
    double b[] = {1.,2.,3.};
    double dot,mag_a,mag_b;
    double  *d_a, *d_b, *d_o;
    int size= 3* sizeof(double);

    hipStream_t stream1,stream2,stream3;
    // hipError_t flag;

    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);
    hipStreamCreate(&stream3);

    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_o, size);

    hipMemcpyAsync(&d_a, &a, size, hipMemcpyHostToDevice,stream1);
    hipMemcpyAsync(&d_b, &b, size, hipMemcpyHostToDevice,stream1);
    dot<<<nBlocks,nThreads,0,stream1>>>(d_a,d_b,d_o);
    hipMemcpyAsync(&dot, &d_o, sizeof(double), hipMemcpyDeviceToHost, stream1);

    hipMemcpyAsync(&d_a, a, size, hipMemcpyHostToDevice,stream2);
    magnitude<<<nBlocks,nThreads,0,stream2>>>(d_a,d_o);
    hipMemcpyAsync(&mag_a, &d_o, sizeof(double), hipMemcpyDeviceToHost, stream2);

    hipMemcpyAsync(&d_b, b, size, hipMemcpyHostToDevice,stream3);
    magnitude<<<nBlocks,nThreads,0,stream3>>>(d_b,d_o);
    hipMemcpyAsync(&mag_b, &d_o, sizeof(double), hipMemcpyDeviceToHost, stream3);

    hipDeviceSynchronize();
    
    result = dot/(mag_a*mag_b);
    std::cout<<"Result: "<<result<<std::endl;
    // hipFree(d_a); hipFree(d_b); hipFree(d_o);
    
    return 0;
}