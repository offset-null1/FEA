#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>
#include <opencv2/highgui.hpp>
#include <math.h>

#define TILE_SIZE 4

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
    if(code != hipSuccess){
        std::cerr << "GPU assert:" << file <<":"<< line<<":"<< hipGetErrorString(code) <<"\n";
        if(abort) exit(code);
    }
}

__global__ void kernel(unsigned char *in, unsigned char* out, uint frameWidth, uint frameHeight){

   uint col = blockDim.x * blockIdx.x + threadIdx.x;
   uint row = blockDim.y * blockIdx.y + threadIdx.y;

    __shared__ unsigned char window[(TILE_SIZE + 2)][(TILE_SIZE + 2)];

    bool is_xleft = (threadIdx.x == 0) , is_xright = (threadIdx.x == TILE_SIZE - 1);
    bool is_ytop = (threadIdx.y == 0) , is_ybottom = (threadIdx.y == TILE_SIZE - 1);

    //initialize boundary conditions (for all tiles) padding = 0

    if(is_xleft)  { window[threadIdx.x][threadIdx.y + 1] = 0; }

    else if(is_xright) { window[threadIdx.x + 2][threadIdx.y + 1] = 0; }
    
    if(is_ytop)   { window[threadIdx.x + 1][threadIdx.y] = 0; 

        if(is_xleft) { window[threadIdx.x][threadIdx.y] = 0; }
        else if(is_xright) { window[threadIdx.x + 2][threadIdx.y] = 0; }
    }
   
    else if(is_ybottom) { window[threadIdx.x + 1][threadIdx.y + 2] = 0; 

        if(is_xleft) { window[threadIdx.x][threadIdx.y + 2] = 0; }
        else if(is_xright) { window[threadIdx.x + 2][threadIdx.y + 2] = 0; }
    }
    
    //store frame values except paddings 
    window[threadIdx.x + 1][threadIdx.y + 1] = in[((row * frameWidth)) + col];
    
    //check if middle tiles then populate the padding values
    if(is_xleft && (col>0)) { window[threadIdx.x][threadIdx.y + 1] = in[(row * frameWidth) + (col-1)]; }

    else if(is_xright && (col<(frameWidth-1))) { window[threadIdx.x + 2][threadIdx.y + 1] = in[(row * frameWidth) + (col+1)]; }

    if(is_ytop && (row>0)){
        window[threadIdx.x + 1][threadIdx.y] = in[((row-1) * frameWidth) + col];

        if(is_xleft) { window[threadIdx.x][threadIdx.y] = in[((row-1) * frameWidth) + (col-1)]; }
        else if(is_xright) { window[threadIdx.x+2][threadIdx.y] = in[((row-1) * frameWidth) + (col+1)]; }

    } 

    else if(is_ybottom && row<(frameHeight-1)){
        window[threadIdx.x + 1][threadIdx.y + 2] = in[((row+1) * frameWidth) + col];

        if(is_xleft) {window[threadIdx.x][threadIdx.y + 2] = in[((row+1) * frameWidth) + (col-1)]; }
        else if(is_xright) {window[threadIdx.x+2][threadIdx.y + 2] = in[((row+1) * frameWidth) + (col+1)]; }

    } 

    __syncthreads();

    //end storing

    //filter setup
    if(col<(frameWidth-1) && row<(frameHeight-1)){
        unsigned char filterWindow[9] = { window[threadIdx.x][threadIdx.y]    , window[threadIdx.x + 1][threadIdx.y]    , window[threadIdx.x+2][threadIdx.y]      ,
            window[threadIdx.x][threadIdx.y + 1], window[threadIdx.x + 1][threadIdx.y + 1], window[threadIdx.x + 2][threadIdx.y + 1],
            window[threadIdx.x][threadIdx.y + 2], window[threadIdx.x + 1][threadIdx.y + 2], window[threadIdx.x + 2][threadIdx.y + 2]  };

//sort
    
        for(uint i=0 ; i<9 ; ++i){
            for(uint j=i+1 ; j<9 ; ++j){
                if(filterWindow[i] > filterWindow[j]){
                    unsigned char temp = filterWindow[i];
                    filterWindow[i]    = filterWindow[j];
                    filterWindow[j]    = temp;
                }
            }
        }
    
        out[(row * frameWidth) + col] = filterWindow[4]; //store median
    }

}


extern "C"{

    void take_input(const cv::Mat& in_frame, const cv::Mat& out_frame){
        unsigned char* in;
        unsigned char* out;
        uint frameWidth = in_frame.cols;
        uint frameHeight = in_frame.rows;
    
        size_t d_ipSize = in_frame.cols * in_frame.rows;
        size_t d_outSize = (in_frame.cols/2) * (in_frame.rows/2);
    
        hipEvent_t start,stop;
    
        gpuErrchk( hipEventCreate(&start) );
        gpuErrchk( hipEventCreate(&stop) );
        gpuErrchk( hipEventRecord(start) );
      
        gpuErrchk( hipMalloc( (void**)& in, d_ipSize) );
        gpuErrchk( hipMalloc( (void**)& out, d_ipSize) );

        gpuErrchk( hipMemcpy( in, in_frame.data, d_ipSize, hipMemcpyHostToDevice) );                  
   
        dim3 threads(TILE_SIZE, TILE_SIZE);                                                                  
        dim3 blocks( (in_frame.cols /threads.x)+1 , (in_frame.rows /threads.y)+1 );
    
        kernel <<< blocks,threads >>> (in, out, frameWidth, frameHeight);
    
        gpuErrchk( hipDeviceSynchronize() );
    
        gpuErrchk( hipMemcpy( out_frame.data, out, d_ipSize, hipMemcpyDeviceToHost ) );
    
        hipFree(in);
        hipFree(out);
    
        gpuErrchk( hipEventRecord(stop) );
        float time = 0;
        hipEventElapsedTime(&time, start, stop);
    
        // std::cout<<"Time :"<<time<<"\n";
        printf("time %f\n", time); 
       
    }
}
